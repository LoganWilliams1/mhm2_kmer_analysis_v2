#include "gpu_alns.hpp"
#include"utils_gpu.hpp"
gpu_alignments::gpu_alignments(int max_alignments){
    cudaErrchk(hipMalloc(&offset_query_gpu, (max_alignments) * sizeof(int)));
    cudaErrchk(hipMalloc(&offset_ref_gpu, (max_alignments) * sizeof(int)));
    cudaErrchk(hipMalloc(&ref_start_gpu, (max_alignments) * sizeof(short)));
    cudaErrchk(hipMalloc(&ref_end_gpu, (max_alignments) * sizeof(short)));
    cudaErrchk(hipMalloc(&query_start_gpu, (max_alignments) * sizeof(short)));
    cudaErrchk(hipMalloc(&query_end_gpu, (max_alignments) * sizeof(short)));
    cudaErrchk(hipMalloc(&scores_gpu, (max_alignments) * sizeof(short)));
}

gpu_alignments::~gpu_alignments(){
    cudaErrchk(hipFree(offset_query_gpu)); offset_query_gpu = nullptr;
    cudaErrchk(hipFree(offset_ref_gpu));   offset_ref_gpu = nullptr;
    cudaErrchk(hipFree(ref_start_gpu));    ref_start_gpu = nullptr;
    cudaErrchk(hipFree(ref_end_gpu));      ref_end_gpu = nullptr;
    cudaErrchk(hipFree(query_start_gpu));  query_start_gpu = nullptr;
    cudaErrchk(hipFree(query_end_gpu));    query_end_gpu = nullptr;
    cudaErrchk(hipFree(scores_gpu));       scores_gpu = nullptr;
}